#include "hip/hip_runtime.h"
#include "Logging.h"

#include "Core.cuh"
//#include "Scene.h"
//#include "helper_math.cuh"
#include "Camera2.cuh"
#include "Camera2Impl.cuh"
#include "Lighting2.cuh"
#include "Lighting2Impl.cuh"
#include "DenoiseParams.cuh"

CD float3		gAaBbMin;
CD float3		gAaBbMax;
CD float3		gInvAaBbMin;
CD float3		gInvAaBbMax;
CD float		gStepSize;
CD float		gStepSizeShadow;
CD float		gDensityScale;
CD float		gGradientDelta;
CD float		gInvGradientDelta;
CD float3		gGradientDeltaX;
CD float3		gGradientDeltaY;
CD float3		gGradientDeltaZ;
CD int			gFilmWidth;
CD int			gFilmHeight;
CD int			gFilmNoPixels;
CD int			gFilterWidth;
CD float		gFilterWeights[10];
CD float		gExposure;
CD float		gInvExposure;
CD float		gGamma;
CD float		gInvGamma;
CD float		gDenoiseEnabled;
CD int		gDenoiseWindowRadius;
CD float		gDenoiseInvWindowArea;
CD float		gDenoiseNoise;
CD float		gDenoiseWeightThreshold;
CD float		gDenoiseLerpThreshold;
CD float		gDenoiseLerpC;
CD float		gNoIterations;
CD float		gInvNoIterations;

CD int gShadingType;
CD float gGradientFactor;

CD CudaLighting gLighting;

// enough data to generate a camera ray
CD CudaCamera gCamera;

#define TF_NO_SAMPLES		128
#define INV_TF_NO_SAMPLES	1.0f / (float)TF_NO_SAMPLES

//#include "Camera.cuh"
//#include "Model.cuh"
//#include "View.cuh"
//#include "Blur.cuh"
#include "Denoise.cuh"
#include "Estimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "NearestIntersection.cuh"
//#include "SpecularBloom.cuh"
#include "ToneMap.cuh"

void RGBToFloat3(const CColorRgbHdr* src, float3* dest) {
	dest->x = src->r;
	dest->y = src->g;
	dest->z = src->b;
}


void BindConstants(const CudaLighting& cudalt, const CDenoiseParams& denoise, const CudaCamera& cudacam, 
	const CBoundingBox& bbox, const CRenderSettings& renderSettings, int numIterations,
	int w, int h, float gamma, float exposure)
{
	const float3 AaBbMin = make_float3(bbox.GetMinP().x, bbox.GetMinP().y, bbox.GetMinP().z);
	const float3 AaBbMax = make_float3(bbox.GetMaxP().x, bbox.GetMaxP().y, bbox.GetMaxP().z);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gAaBbMin), &AaBbMin, sizeof(float3)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gAaBbMax), &AaBbMax, sizeof(float3)));

	const float3 InvAaBbMin = make_float3(bbox.GetInvMinP().x, bbox.GetInvMinP().y, bbox.GetInvMinP().z);
	const float3 InvAaBbMax = make_float3(bbox.GetInvMaxP().x, bbox.GetInvMaxP().y, bbox.GetInvMaxP().z);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvAaBbMin), &InvAaBbMin, sizeof(float3)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvAaBbMax), &InvAaBbMax, sizeof(float3)));

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gShadingType), &renderSettings.m_ShadingType, sizeof(int)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGradientFactor), &renderSettings.m_GradientFactor, sizeof(float)));

	const float StepSize		= renderSettings.m_StepSizeFactor * renderSettings.m_GradientDelta;
	const float StepSizeShadow	= renderSettings.m_StepSizeFactorShadow * renderSettings.m_GradientDelta;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gStepSize), &StepSize, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gStepSizeShadow), &StepSizeShadow, sizeof(float)));

	const float DensityScale = renderSettings.m_DensityScale;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDensityScale), &DensityScale, sizeof(float)));
	
	const float GradientDelta		= 1.0f * renderSettings.m_GradientDelta;
	const float InvGradientDelta	= 1.0f / GradientDelta;
	const Vec3f GradientDeltaX(GradientDelta, 0.0f, 0.0f);
	const Vec3f GradientDeltaY(0.0f, GradientDelta, 0.0f);
	const Vec3f GradientDeltaZ(0.0f, 0.0f, GradientDelta);
	
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGradientDelta), &GradientDelta, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvGradientDelta), &InvGradientDelta, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGradientDeltaX), &GradientDeltaX, sizeof(float3)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGradientDeltaY), &GradientDeltaY, sizeof(float3)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGradientDeltaZ), &GradientDeltaZ, sizeof(float3)));
	
	const int FilmWidth		= w;
	const int Filmheight	= h;
	//const int FilmNoPixels	= camera.m_Film.m_Resolution.GetNoElements();

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gFilmWidth), &FilmWidth, sizeof(int)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gFilmHeight), &Filmheight, sizeof(int)));
	//HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gFilmNoPixels), &FilmNoPixels, sizeof(int)));

	const int FilterWidth = 1;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gFilterWidth), &FilterWidth, sizeof(int)));

	const float FilterWeights[10] = { 0.11411459588254977f, 0.08176668094332218f, 0.03008028089187349f, 0.01f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gFilterWeights), FilterWeights, 10 * sizeof(float)));

	const float Gamma		= gamma;
	const float InvGamma	= 1.0f / Gamma;
	const float Exposure	= exposure;
	const float InvExposure	= 1.0f / Exposure;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gExposure), &Exposure, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvExposure), &InvExposure, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGamma), &Gamma, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvGamma), &InvGamma, sizeof(float)));

	const float denoiseEnabled = denoise.m_Enabled ? 1.0f : 0.0f;
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseEnabled), &denoiseEnabled, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseWindowRadius), &denoise.m_WindowRadius, sizeof(int)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseInvWindowArea), &denoise.m_InvWindowArea, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseNoise), &denoise.m_Noise, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseWeightThreshold), &denoise.m_WeightThreshold, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseLerpThreshold), &denoise.m_LerpThreshold, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseLerpC), &denoise.m_LerpC, sizeof(float)));

	const float NoIterations	= numIterations;
	const float InvNoIterations = 1.0f / ((NoIterations > 1.0f) ? NoIterations : 1.0f);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gNoIterations), &NoIterations, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvNoIterations), &InvNoIterations, sizeof(float)));

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gCamera), &cudacam, sizeof(CudaCamera)));

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gLighting), &cudalt, sizeof(CudaLighting)));
}

// BindConstants must be called first to initialize vars used by kernels
void Render(const int& Type, int numExposures, int w, int h,
	cudaFB& framebuffers,
	const cudaVolume& volumedata,
	CTiming& RenderImage, CTiming& BlurImage, CTiming& PostProcessImage, CTiming& DenoiseImage,
	int& numIterations)
{
	// find nearest intersection to set camera focal distance automatically.
	// then re-upload that data.
	//if (camera.m_Focus.m_Type == 0) {
		float fd = NearestIntersection(volumedata);
		//camera.m_Focus.m_FocalDistance = NearestIntersection(volumedata);
		// send m_FocalDistance back to gpu.
		//CudaCamera c;
		//FillCudaCamera(&camera, c);
		HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gCamera), &fd, sizeof(float)));
	//}

	for (int i = 0; i < numExposures; ++i) {
		CCudaTimer TmrRender;

		switch (Type)
		{
		case 0:
		{
			SingleScattering(w, h,
				volumedata, framebuffers.fb, framebuffers.randomSeeds1, framebuffers.randomSeeds2);
			break;
		}

		case 1:
		{
			//			MultipleScattering(&Scene);
			break;
		}
		}
		RenderImage.AddDuration(TmrRender.ElapsedTime());

		// estimate just adds to accumulation buffer.
		CCudaTimer TmrPostProcess;
		Estimate(w, h, 
			framebuffers.fb, framebuffers.fbaccum);
		PostProcessImage.AddDuration(TmrPostProcess.ElapsedTime());

		numIterations++;
		const float NoIterations = numIterations;
		const float InvNoIterations = 1.0f / ((NoIterations > 1.0f) ? NoIterations : 1.0f);
		HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gNoIterations), &NoIterations, sizeof(float)));
		HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvNoIterations), &InvNoIterations, sizeof(float)));
	}
}
