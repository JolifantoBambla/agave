#include "hip/hip_runtime.h"
#include "Logging.h"

#include "Core.cuh"

//texture<short, hipTextureType3D, hipReadModeNormalizedFloat>		gTexDensity;
//texture<short, hipTextureType3D, hipReadModeNormalizedFloat>		gTexGradientMagnitude;

hipTextureObject_t gTexDensity;
hipTextureObject_t gTexGradientMagnitude;

//texture<float, hipTextureType3D, hipReadModeElementType>			gTexExtinction;
//texture<float, hipTextureType1D, hipReadModeElementType>			gTexOpacity;
//texture<float4, hipTextureType1D, hipReadModeElementType>			gTexDiffuse;
//texture<float4, hipTextureType1D, hipReadModeElementType>			gTexSpecular;
//texture<float, hipTextureType1D, hipReadModeElementType>			gTexRoughness;
//texture<float4, hipTextureType1D, hipReadModeElementType>			gTexEmission;
texture<uchar4, hipTextureType2D, hipReadModeNormalizedFloat>		gTexRunningEstimateRgba;

hipArray* gpDensityArray				= NULL;
hipArray* gpGradientMagnitudeArray		= NULL;
//hipArray* gpOpacityArray				= NULL;
//hipArray* gpDiffuseArray				= NULL;
//hipArray* gpSpecularArray				= NULL;
//hipArray* gpRoughnessArray				= NULL;
//hipArray* gpEmissionArray				= NULL;

CD float3		gAaBbMin;
CD float3		gAaBbMax;
CD float3		gInvAaBbMin;
CD float3		gInvAaBbMax;
CD float		gIntensityMin;
CD float		gIntensityMax;
CD float		gIntensityRange;
CD float		gIntensityInvRange;
CD float		gStepSize;
CD float		gStepSizeShadow;
CD float		gDensityScale;
CD float		gGradientDelta;
CD float		gInvGradientDelta;
CD float3		gGradientDeltaX;
CD float3		gGradientDeltaY;
CD float3		gGradientDeltaZ;
CD int			gFilmWidth;
CD int			gFilmHeight;
CD int			gFilmNoPixels;
CD int			gFilterWidth;
CD float		gFilterWeights[10];
CD float		gExposure;
CD float		gInvExposure;
CD float		gGamma;
CD float		gInvGamma;
CD float		gDenoiseEnabled;
CD int		gDenoiseWindowRadius;
CD float		gDenoiseInvWindowArea;
CD float		gDenoiseNoise;
CD float		gDenoiseWeightThreshold;
CD float		gDenoiseLerpThreshold;
CD float		gDenoiseLerpC;
CD float		gNoIterations;
CD float		gInvNoIterations;

CD float4		gDiffuseColor;
CD float4		gSpecularColor;
CD float4		gEmissiveColor;

#define TF_NO_SAMPLES		128
#define INV_TF_NO_SAMPLES	1.0f / (float)TF_NO_SAMPLES

//#include "Model.cuh"
#include "View.cuh"
#include "Blur.cuh"
#include "Denoise.cuh"
#include "Estimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "NearestIntersection.cuh"
//#include "SpecularBloom.cuh"
#include "ToneMap.cuh"

CCudaView	gRenderCanvasView;

void BindDensityBuffer(short* pBuffer, hipExtent volumeSize)
{
	// create 3D array
	hipChannelFormatDesc gradientChannelDesc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);
	HandleCudaError(hipMalloc3DArray(&gpDensityArray, &gradientChannelDesc, volumeSize));

	// copy data to 3D array
	hipMemcpy3DParms gradientCopyParams = { 0 };
	gradientCopyParams.srcPtr = make_hipPitchedPtr(pBuffer, volumeSize.width*sizeof(short), volumeSize.width, volumeSize.height);
	gradientCopyParams.dstArray = gpDensityArray;
	gradientCopyParams.extent = volumeSize;
	gradientCopyParams.kind = hipMemcpyHostToDevice;
	HandleCudaError(hipMemcpy3D(&gradientCopyParams));

	hipResourceDesc gradientTexRes;
	memset(&gradientTexRes, 0, sizeof(hipResourceDesc));
	gradientTexRes.resType = hipResourceTypeArray;
	gradientTexRes.res.array.array = gpDensityArray;
	hipTextureDesc     gradientTexDescr;
	memset(&gradientTexDescr, 0, sizeof(hipTextureDesc));
	gradientTexDescr.normalizedCoords = 1;
	gradientTexDescr.filterMode = hipFilterModeLinear;
	gradientTexDescr.addressMode[0] = hipAddressModeClamp;   // clamp
	gradientTexDescr.addressMode[1] = hipAddressModeClamp;
	gradientTexDescr.addressMode[2] = hipAddressModeClamp;
	gradientTexDescr.readMode = hipReadModeNormalizedFloat;
	HandleCudaError(hipCreateTextureObject(&gTexDensity, &gradientTexRes, &gradientTexDescr, NULL));
}

void BindGradientMagnitudeBuffer(short* pBuffer, hipExtent volumeSize)
{
	// create 3D array
	hipChannelFormatDesc gradientChannelDesc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);
	HandleCudaError(hipMalloc3DArray(&gpGradientMagnitudeArray, &gradientChannelDesc, volumeSize));

	// copy data to 3D array
	hipMemcpy3DParms gradientCopyParams = { 0 };
	gradientCopyParams.srcPtr = make_hipPitchedPtr(pBuffer, volumeSize.width*sizeof(short), volumeSize.width, volumeSize.height);
	gradientCopyParams.dstArray = gpGradientMagnitudeArray;
	gradientCopyParams.extent = volumeSize;
	gradientCopyParams.kind = hipMemcpyHostToDevice;
	HandleCudaError(hipMemcpy3D(&gradientCopyParams));

	hipResourceDesc gradientTexRes;
	memset(&gradientTexRes, 0, sizeof(hipResourceDesc));
	gradientTexRes.resType = hipResourceTypeArray;
	gradientTexRes.res.array.array = gpGradientMagnitudeArray;
	hipTextureDesc     gradientTexDescr;
	memset(&gradientTexDescr, 0, sizeof(hipTextureDesc));
	gradientTexDescr.normalizedCoords = 1;
	gradientTexDescr.filterMode = hipFilterModeLinear;
	gradientTexDescr.addressMode[0] = hipAddressModeClamp;   // clamp
	gradientTexDescr.addressMode[1] = hipAddressModeClamp;
	gradientTexDescr.addressMode[2] = hipAddressModeClamp;
	gradientTexDescr.readMode = hipReadModeNormalizedFloat;
	HandleCudaError(hipCreateTextureObject(&gTexGradientMagnitude, &gradientTexRes, &gradientTexDescr, NULL));
}

void UnbindDensityBuffer(void)
{
	HandleCudaError(hipFreeArray(gpDensityArray));
	gpDensityArray = NULL;
	HandleCudaError(hipDestroyTextureObject(gTexDensity));
}

void UnbindGradientMagnitudeBuffer(void)
{
	HandleCudaError(hipFreeArray(gpGradientMagnitudeArray));
	gpGradientMagnitudeArray = NULL;
	HandleCudaError(hipDestroyTextureObject(gTexGradientMagnitude));
}

void BindRenderCanvasView(const CResolution2D& Resolution)
{
	gRenderCanvasView.Resize(Resolution);

	hipChannelFormatDesc Channel;
	
	Channel = hipCreateChannelDesc<uchar4>();

	HandleCudaError(hipBindTexture2D(0, gTexRunningEstimateRgba, gRenderCanvasView.m_EstimateRgbaLdr.GetPtr(), Channel, gRenderCanvasView.GetWidth(), gRenderCanvasView.GetHeight(), gRenderCanvasView.m_EstimateRgbaLdr.GetPitch()));
}

void ResetRenderCanvasView(void)
{
	gRenderCanvasView.Reset();
}

void FreeRenderCanvasView(void)
{
	gRenderCanvasView.Free();
}

unsigned char* GetDisplayEstimate(void)
{
	return (unsigned char*)gRenderCanvasView.m_DisplayEstimateRgbLdr.GetPtr(0, 0);
}
#if 0
void BindTransferFunctionOpacity(CTransferFunction& TransferFunctionOpacity)
{
	gTexOpacity.normalized		= true;
	gTexOpacity.filterMode		= hipFilterModeLinear;
	gTexOpacity.addressMode[0]	= hipAddressModeClamp;

	float Opacity[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
		Opacity[i] = TransferFunctionOpacity.F((float)i * INV_TF_NO_SAMPLES).r;
	
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float>();

	if (gpOpacityArray == NULL)
		HandleCudaError(hipMallocArray(&gpOpacityArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpOpacityArray, 0, 0, Opacity, TF_NO_SAMPLES * sizeof(float), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexOpacity, gpOpacityArray, ChannelDesc));
}

void UnbindTransferFunctionOpacity(void)
{
	HandleCudaError(hipFreeArray(gpOpacityArray));
	gpOpacityArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexOpacity));
}

void BindTransferFunctionDiffuse(CTransferFunction& TransferFunctionDiffuse)
{
	gTexDiffuse.normalized		= true;
	gTexDiffuse.filterMode		= hipFilterModeLinear;
	gTexDiffuse.addressMode[0]	= hipAddressModeClamp;

	float4 Diffuse[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
	{
		Diffuse[i].x = TransferFunctionDiffuse.F((float)i * INV_TF_NO_SAMPLES).r;
		Diffuse[i].y = TransferFunctionDiffuse.F((float)i * INV_TF_NO_SAMPLES).g;
		Diffuse[i].z = TransferFunctionDiffuse.F((float)i * INV_TF_NO_SAMPLES).b;
	}

	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float4>();
	
	if (gpDiffuseArray == NULL)
		HandleCudaError(hipMallocArray(&gpDiffuseArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpDiffuseArray, 0, 0, Diffuse, TF_NO_SAMPLES * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexDiffuse, gpDiffuseArray, ChannelDesc));
}

void UnbindTransferFunctionDiffuse(void)
{
	HandleCudaError(hipFreeArray(gpDiffuseArray));
	gpDiffuseArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexDiffuse));
}

void BindTransferFunctionSpecular(CTransferFunction& TransferFunctionSpecular)
{
	gTexSpecular.normalized		= true;
	gTexSpecular.filterMode		= hipFilterModeLinear;
	gTexSpecular.addressMode[0]	= hipAddressModeClamp;

	float4 Specular[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
	{
		Specular[i].x = TransferFunctionSpecular.F((float)i * INV_TF_NO_SAMPLES).r;
		Specular[i].y = TransferFunctionSpecular.F((float)i * INV_TF_NO_SAMPLES).g;
		Specular[i].z = TransferFunctionSpecular.F((float)i * INV_TF_NO_SAMPLES).b;
	}

	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float4>();
	
	if (gpSpecularArray == NULL)
		HandleCudaError(hipMallocArray(&gpSpecularArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpSpecularArray, 0, 0, Specular, TF_NO_SAMPLES * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexSpecular, gpSpecularArray, ChannelDesc));
}

void UnbindTransferFunctionSpecular(void)
{
	HandleCudaError(hipFreeArray(gpSpecularArray));
	gpSpecularArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexSpecular));
}

void BindTransferFunctionRoughness(CTransferFunction& TransferFunctionRoughness)
{
	gTexRoughness.normalized		= true;
	gTexRoughness.filterMode		= hipFilterModeLinear;
	gTexRoughness.addressMode[0]	= hipAddressModeClamp;

	float Roughness[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
		Roughness[i] = TransferFunctionRoughness.F((float)i * INV_TF_NO_SAMPLES).r;
	
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float>();

	if (gpRoughnessArray == NULL)
		HandleCudaError(hipMallocArray(&gpRoughnessArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpRoughnessArray, 0, 0, Roughness, TF_NO_SAMPLES * sizeof(float),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexRoughness, gpRoughnessArray, ChannelDesc));
}

void UnbindTransferFunctionRoughness(void)
{
	HandleCudaError(hipFreeArray(gpRoughnessArray));
	gpRoughnessArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexRoughness));
}

void BindTransferFunctionEmission(CTransferFunction& TransferFunctionEmission)
{
	gTexEmission.normalized		= true;
	gTexEmission.filterMode		= hipFilterModeLinear;
	gTexEmission.addressMode[0]	= hipAddressModeClamp;

	float4 Emission[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
	{
		Emission[i].x = TransferFunctionEmission.F((float)i * INV_TF_NO_SAMPLES).r;
		Emission[i].y = TransferFunctionEmission.F((float)i * INV_TF_NO_SAMPLES).g;
		Emission[i].z = TransferFunctionEmission.F((float)i * INV_TF_NO_SAMPLES).b;
	}

	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float4>();
	
	if (gpEmissionArray == NULL)
		HandleCudaError(hipMallocArray(&gpEmissionArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpEmissionArray, 0, 0, Emission, TF_NO_SAMPLES * sizeof(float4),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexEmission, gpEmissionArray, ChannelDesc));
}

void UnbindTransferFunctionEmission(void)
{
	HandleCudaError(hipFreeArray(gpEmissionArray));
	gpEmissionArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexEmission));
}
#endif
void BindConstants(CScene* pScene)
{
	const float3 AaBbMin = make_float3(pScene->m_BoundingBox.GetMinP().x, pScene->m_BoundingBox.GetMinP().y, pScene->m_BoundingBox.GetMinP().z);
	const float3 AaBbMax = make_float3(pScene->m_BoundingBox.GetMaxP().x, pScene->m_BoundingBox.GetMaxP().y, pScene->m_BoundingBox.GetMaxP().z);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gAaBbMin), &AaBbMin, sizeof(float3)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gAaBbMax), &AaBbMax, sizeof(float3)));

	const float3 InvAaBbMin = make_float3(pScene->m_BoundingBox.GetInvMinP().x, pScene->m_BoundingBox.GetInvMinP().y, pScene->m_BoundingBox.GetInvMinP().z);
	const float3 InvAaBbMax = make_float3(pScene->m_BoundingBox.GetInvMaxP().x, pScene->m_BoundingBox.GetInvMaxP().y, pScene->m_BoundingBox.GetInvMaxP().z);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvAaBbMin), &InvAaBbMin, sizeof(float3)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvAaBbMax), &InvAaBbMax, sizeof(float3)));

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDiffuseColor), pScene->m_DiffuseColor, sizeof(float4)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gSpecularColor), pScene->m_SpecularColor, sizeof(float4)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gEmissiveColor), pScene->m_EmissiveColor, sizeof(float4)));

	const float IntensityMin		= pScene->m_IntensityRange.GetMin();
	const float IntensityMax		= pScene->m_IntensityRange.GetMax();
	const float IntensityRange		= pScene->m_IntensityRange.GetRange();
	const float IntensityInvRange	= 1.0f / IntensityRange;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gIntensityMin), &IntensityMin, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gIntensityMax), &IntensityMax, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gIntensityRange), &IntensityRange, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gIntensityInvRange), &IntensityInvRange, sizeof(float)));

	const float StepSize		= pScene->m_StepSizeFactor * pScene->m_GradientDelta;
	const float StepSizeShadow	= pScene->m_StepSizeFactorShadow * pScene->m_GradientDelta;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gStepSize), &StepSize, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gStepSizeShadow), &StepSizeShadow, sizeof(float)));

	const float DensityScale = pScene->m_DensityScale;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDensityScale), &DensityScale, sizeof(float)));
	
	const float GradientDelta		= 1.0f * pScene->m_GradientDelta;
	const float InvGradientDelta	= 1.0f / GradientDelta;
	const Vec3f GradientDeltaX(GradientDelta, 0.0f, 0.0f);
	const Vec3f GradientDeltaY(0.0f, GradientDelta, 0.0f);
	const Vec3f GradientDeltaZ(0.0f, 0.0f, GradientDelta);
	
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGradientDelta), &GradientDelta, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvGradientDelta), &InvGradientDelta, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGradientDeltaX), &GradientDeltaX, sizeof(float3)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGradientDeltaY), &GradientDeltaY, sizeof(float3)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGradientDeltaZ), &GradientDeltaZ, sizeof(float3)));
	
	const int FilmWidth		= pScene->m_Camera.m_Film.GetWidth();
	const int Filmheight	= pScene->m_Camera.m_Film.GetHeight();
	const int FilmNoPixels	= pScene->m_Camera.m_Film.m_Resolution.GetNoElements();

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gFilmWidth), &FilmWidth, sizeof(int)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gFilmHeight), &Filmheight, sizeof(int)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gFilmNoPixels), &FilmNoPixels, sizeof(int)));

	const int FilterWidth = 1;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gFilterWidth), &FilterWidth, sizeof(int)));

	const float FilterWeights[10] = { 0.11411459588254977f, 0.08176668094332218f, 0.03008028089187349f, 0.01f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gFilterWeights), FilterWeights, 10 * sizeof(float)));

	const float Gamma		= pScene->m_Camera.m_Film.m_Gamma;
	const float InvGamma	= 1.0f / Gamma;
	const float Exposure	= pScene->m_Camera.m_Film.m_Exposure;
	const float InvExposure	= 1.0f / Exposure;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gExposure), &Exposure, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvExposure), &InvExposure, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGamma), &Gamma, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvGamma), &InvGamma, sizeof(float)));

	const float denoiseEnabled = pScene->m_DenoiseParams.m_Enabled ? 1.0f : 0.0f;
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseEnabled), &denoiseEnabled, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseWindowRadius), &pScene->m_DenoiseParams.m_WindowRadius, sizeof(int)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseInvWindowArea), &pScene->m_DenoiseParams.m_InvWindowArea, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseNoise), &pScene->m_DenoiseParams.m_Noise, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseWeightThreshold), &pScene->m_DenoiseParams.m_WeightThreshold, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseLerpThreshold), &pScene->m_DenoiseParams.m_LerpThreshold, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseLerpC), &pScene->m_DenoiseParams.m_LerpC, sizeof(float)));

	const float NoIterations	= pScene->GetNoIterations();
	const float InvNoIterations = 1.0f / ((NoIterations > 1.0f) ? NoIterations : 1.0f);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gNoIterations), &NoIterations, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvNoIterations), &InvNoIterations, sizeof(float)));
}

void Render(const int& Type, CScene& Scene,
	cudaFB& framebuffers,
	const cudaVolume& volumedata,
	CTiming& RenderImage, CTiming& BlurImage, CTiming& PostProcessImage, CTiming& DenoiseImage)
{

	//HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gTexDensity), &volumedata.volumeTexture, sizeof(hipTextureObject_t)));
	//HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gTexGradientMagnitude), &volumedata.gradientVolumeTexture, sizeof(hipTextureObject_t)));
	//LOG_DEBUG << "CScene is " << sizeof(CScene) << " bytes";
	CScene* pDevScene = NULL;
	HandleCudaError(hipMalloc(&pDevScene, sizeof(CScene)));
	// copy entire Scene(host mem) up to gpu.
	HandleCudaError(hipMemcpy(pDevScene, &Scene, sizeof(CScene), hipMemcpyHostToDevice));
	if (Scene.m_Camera.m_Focus.m_Type == 0) {
		Scene.m_Camera.m_Focus.m_FocalDistance = NearestIntersection(pDevScene, volumedata);
		HandleCudaError(hipMemcpy(pDevScene, &Scene, sizeof(CScene), hipMemcpyHostToDevice));
	}

	CCudaTimer TmrRender;

	switch (Type)
	{
		case 0:
		{
			SingleScattering(&Scene, pDevScene, volumedata, framebuffers.fb, framebuffers.randomSeeds1, framebuffers.randomSeeds2);
			break;
		}

		case 1:
		{
//			MultipleScattering(&Scene, pDevScene);
			break;
		}
	}
	RenderImage.AddDuration(TmrRender.ElapsedTime());
	
// 	CCudaTimer TmrBlur;
//	Blur(&Scene, pDevScene, pDevView);
//	BlurImage.AddDuration(TmrBlur.ElapsedTime());

	// estimate just adds to accumulation buffer.
	CCudaTimer TmrPostProcess;
	Estimate(&Scene, pDevScene, framebuffers.fb, framebuffers.fbaccum);
	PostProcessImage.AddDuration(TmrPostProcess.ElapsedTime());

//	Scene.SetNoIterations(Scene.GetNoIterations() + 1);

//	HandleCudaError(hipMemcpy(pDevScene, &Scene, sizeof(CScene), hipMemcpyHostToDevice));
//	SingleScattering(&Scene, pDevScene, volumedata, framebuffers.fb, framebuffers.randomSeeds1, framebuffers.randomSeeds2);
//	Estimate(&Scene, pDevScene, framebuffers.fb, framebuffers.fbaccum);

//	Scene.SetNoIterations(Scene.GetNoIterations() + 1);

	// tone map to prep for display

	//ToneMap(float* inbuf, hipSurfaceObject_t surfaceObj, int w, int h)

//	CCudaTimer TmrDenoise;
//	Denoise(&Scene, pDevScene, pDevView);
//	DenoiseImage.AddDuration(TmrDenoise.ElapsedTime());
	

	HandleCudaError(hipFree(pDevScene));
	pDevScene = NULL;

}