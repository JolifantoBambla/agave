#include "hip/hip_runtime.h"
#include "Logging.h"

#include "Core.cuh"
#include "helper_math.cuh"
#include "Camera2.cuh"
#include "Lighting2.cuh"

//texture<short, hipTextureType3D, hipReadModeNormalizedFloat>		gTexDensity;
//texture<short, hipTextureType3D, hipReadModeNormalizedFloat>		gTexGradientMagnitude;

hipTextureObject_t gTexDensity;
hipTextureObject_t gTexGradientMagnitude;

//texture<float, hipTextureType3D, hipReadModeElementType>			gTexExtinction;
//texture<float, hipTextureType1D, hipReadModeElementType>			gTexOpacity;
//texture<float4, hipTextureType1D, hipReadModeElementType>			gTexDiffuse;
//texture<float4, hipTextureType1D, hipReadModeElementType>			gTexSpecular;
//texture<float, hipTextureType1D, hipReadModeElementType>			gTexRoughness;
//texture<float4, hipTextureType1D, hipReadModeElementType>			gTexEmission;
texture<uchar4, hipTextureType2D, hipReadModeNormalizedFloat>		gTexRunningEstimateRgba;

hipArray* gpDensityArray				= NULL;
hipArray* gpGradientMagnitudeArray		= NULL;
//hipArray* gpOpacityArray				= NULL;
//hipArray* gpDiffuseArray				= NULL;
//hipArray* gpSpecularArray				= NULL;
//hipArray* gpRoughnessArray				= NULL;
//hipArray* gpEmissionArray				= NULL;

CD float3		gAaBbMin;
CD float3		gAaBbMax;
CD float3		gInvAaBbMin;
CD float3		gInvAaBbMax;
CD float		gIntensityMin;
CD float		gIntensityMax;
CD float		gIntensityRange;
CD float		gIntensityInvRange;
CD float		gStepSize;
CD float		gStepSizeShadow;
CD float		gDensityScale;
CD float		gGradientDelta;
CD float		gInvGradientDelta;
CD float3		gGradientDeltaX;
CD float3		gGradientDeltaY;
CD float3		gGradientDeltaZ;
CD int			gFilmWidth;
CD int			gFilmHeight;
CD int			gFilmNoPixels;
CD int			gFilterWidth;
CD float		gFilterWeights[10];
CD float		gExposure;
CD float		gInvExposure;
CD float		gGamma;
CD float		gInvGamma;
CD float		gDenoiseEnabled;
CD int		gDenoiseWindowRadius;
CD float		gDenoiseInvWindowArea;
CD float		gDenoiseNoise;
CD float		gDenoiseWeightThreshold;
CD float		gDenoiseLerpThreshold;
CD float		gDenoiseLerpC;
CD float		gNoIterations;
CD float		gInvNoIterations;

CD float4		gDiffuseColor;
CD float4		gSpecularColor;
CD float4		gEmissiveColor;

CD int gShadingType;
CD float gGradientFactor;

CD CudaLighting gLighting;

// enough data to generate a camera ray
CD CudaCamera gCamera;

#define TF_NO_SAMPLES		128
#define INV_TF_NO_SAMPLES	1.0f / (float)TF_NO_SAMPLES

//#include "Model.cuh"
#include "View.cuh"
#include "Blur.cuh"
#include "Denoise.cuh"
#include "Estimate.cuh"
#include "Utilities.cuh"
#include "SingleScattering.cuh"
#include "NearestIntersection.cuh"
//#include "SpecularBloom.cuh"
#include "ToneMap.cuh"

CCudaView	gRenderCanvasView;

void BindDensityBuffer(short* pBuffer, hipExtent volumeSize)
{
	// create 3D array
	hipChannelFormatDesc gradientChannelDesc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);
	HandleCudaError(hipMalloc3DArray(&gpDensityArray, &gradientChannelDesc, volumeSize));

	// copy data to 3D array
	hipMemcpy3DParms gradientCopyParams = { 0 };
	gradientCopyParams.srcPtr = make_hipPitchedPtr(pBuffer, volumeSize.width*sizeof(short), volumeSize.width, volumeSize.height);
	gradientCopyParams.dstArray = gpDensityArray;
	gradientCopyParams.extent = volumeSize;
	gradientCopyParams.kind = hipMemcpyHostToDevice;
	HandleCudaError(hipMemcpy3D(&gradientCopyParams));

	hipResourceDesc gradientTexRes;
	memset(&gradientTexRes, 0, sizeof(hipResourceDesc));
	gradientTexRes.resType = hipResourceTypeArray;
	gradientTexRes.res.array.array = gpDensityArray;
	hipTextureDesc     gradientTexDescr;
	memset(&gradientTexDescr, 0, sizeof(hipTextureDesc));
	gradientTexDescr.normalizedCoords = 1;
	gradientTexDescr.filterMode = hipFilterModeLinear;
	gradientTexDescr.addressMode[0] = hipAddressModeClamp;   // clamp
	gradientTexDescr.addressMode[1] = hipAddressModeClamp;
	gradientTexDescr.addressMode[2] = hipAddressModeClamp;
	gradientTexDescr.readMode = hipReadModeNormalizedFloat;
	HandleCudaError(hipCreateTextureObject(&gTexDensity, &gradientTexRes, &gradientTexDescr, NULL));
}

void BindGradientMagnitudeBuffer(short* pBuffer, hipExtent volumeSize)
{
	// create 3D array
	hipChannelFormatDesc gradientChannelDesc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);
	HandleCudaError(hipMalloc3DArray(&gpGradientMagnitudeArray, &gradientChannelDesc, volumeSize));

	// copy data to 3D array
	hipMemcpy3DParms gradientCopyParams = { 0 };
	gradientCopyParams.srcPtr = make_hipPitchedPtr(pBuffer, volumeSize.width*sizeof(short), volumeSize.width, volumeSize.height);
	gradientCopyParams.dstArray = gpGradientMagnitudeArray;
	gradientCopyParams.extent = volumeSize;
	gradientCopyParams.kind = hipMemcpyHostToDevice;
	HandleCudaError(hipMemcpy3D(&gradientCopyParams));

	hipResourceDesc gradientTexRes;
	memset(&gradientTexRes, 0, sizeof(hipResourceDesc));
	gradientTexRes.resType = hipResourceTypeArray;
	gradientTexRes.res.array.array = gpGradientMagnitudeArray;
	hipTextureDesc     gradientTexDescr;
	memset(&gradientTexDescr, 0, sizeof(hipTextureDesc));
	gradientTexDescr.normalizedCoords = 1;
	gradientTexDescr.filterMode = hipFilterModeLinear;
	gradientTexDescr.addressMode[0] = hipAddressModeClamp;   // clamp
	gradientTexDescr.addressMode[1] = hipAddressModeClamp;
	gradientTexDescr.addressMode[2] = hipAddressModeClamp;
	gradientTexDescr.readMode = hipReadModeNormalizedFloat;
	HandleCudaError(hipCreateTextureObject(&gTexGradientMagnitude, &gradientTexRes, &gradientTexDescr, NULL));
}

void UnbindDensityBuffer(void)
{
	HandleCudaError(hipFreeArray(gpDensityArray));
	gpDensityArray = NULL;
	HandleCudaError(hipDestroyTextureObject(gTexDensity));
}

void UnbindGradientMagnitudeBuffer(void)
{
	HandleCudaError(hipFreeArray(gpGradientMagnitudeArray));
	gpGradientMagnitudeArray = NULL;
	HandleCudaError(hipDestroyTextureObject(gTexGradientMagnitude));
}

void BindRenderCanvasView(const CResolution2D& Resolution)
{
	gRenderCanvasView.Resize(Resolution);

	hipChannelFormatDesc Channel;
	
	Channel = hipCreateChannelDesc<uchar4>();

	HandleCudaError(hipBindTexture2D(0, gTexRunningEstimateRgba, gRenderCanvasView.m_EstimateRgbaLdr.GetPtr(), Channel, gRenderCanvasView.GetWidth(), gRenderCanvasView.GetHeight(), gRenderCanvasView.m_EstimateRgbaLdr.GetPitch()));
}

void ResetRenderCanvasView(void)
{
	gRenderCanvasView.Reset();
}

void FreeRenderCanvasView(void)
{
	gRenderCanvasView.Free();
}

unsigned char* GetDisplayEstimate(void)
{
	return (unsigned char*)gRenderCanvasView.m_DisplayEstimateRgbLdr.GetPtr(0, 0);
}
#if 0
void BindTransferFunctionOpacity(CTransferFunction& TransferFunctionOpacity)
{
	gTexOpacity.normalized		= true;
	gTexOpacity.filterMode		= hipFilterModeLinear;
	gTexOpacity.addressMode[0]	= hipAddressModeClamp;

	float Opacity[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
		Opacity[i] = TransferFunctionOpacity.F((float)i * INV_TF_NO_SAMPLES).r;
	
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float>();

	if (gpOpacityArray == NULL)
		HandleCudaError(hipMallocArray(&gpOpacityArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpOpacityArray, 0, 0, Opacity, TF_NO_SAMPLES * sizeof(float), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexOpacity, gpOpacityArray, ChannelDesc));
}

void UnbindTransferFunctionOpacity(void)
{
	HandleCudaError(hipFreeArray(gpOpacityArray));
	gpOpacityArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexOpacity));
}

void BindTransferFunctionDiffuse(CTransferFunction& TransferFunctionDiffuse)
{
	gTexDiffuse.normalized		= true;
	gTexDiffuse.filterMode		= hipFilterModeLinear;
	gTexDiffuse.addressMode[0]	= hipAddressModeClamp;

	float4 Diffuse[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
	{
		Diffuse[i].x = TransferFunctionDiffuse.F((float)i * INV_TF_NO_SAMPLES).r;
		Diffuse[i].y = TransferFunctionDiffuse.F((float)i * INV_TF_NO_SAMPLES).g;
		Diffuse[i].z = TransferFunctionDiffuse.F((float)i * INV_TF_NO_SAMPLES).b;
	}

	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float4>();
	
	if (gpDiffuseArray == NULL)
		HandleCudaError(hipMallocArray(&gpDiffuseArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpDiffuseArray, 0, 0, Diffuse, TF_NO_SAMPLES * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexDiffuse, gpDiffuseArray, ChannelDesc));
}

void UnbindTransferFunctionDiffuse(void)
{
	HandleCudaError(hipFreeArray(gpDiffuseArray));
	gpDiffuseArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexDiffuse));
}

void BindTransferFunctionSpecular(CTransferFunction& TransferFunctionSpecular)
{
	gTexSpecular.normalized		= true;
	gTexSpecular.filterMode		= hipFilterModeLinear;
	gTexSpecular.addressMode[0]	= hipAddressModeClamp;

	float4 Specular[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
	{
		Specular[i].x = TransferFunctionSpecular.F((float)i * INV_TF_NO_SAMPLES).r;
		Specular[i].y = TransferFunctionSpecular.F((float)i * INV_TF_NO_SAMPLES).g;
		Specular[i].z = TransferFunctionSpecular.F((float)i * INV_TF_NO_SAMPLES).b;
	}

	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float4>();
	
	if (gpSpecularArray == NULL)
		HandleCudaError(hipMallocArray(&gpSpecularArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpSpecularArray, 0, 0, Specular, TF_NO_SAMPLES * sizeof(float4), hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexSpecular, gpSpecularArray, ChannelDesc));
}

void UnbindTransferFunctionSpecular(void)
{
	HandleCudaError(hipFreeArray(gpSpecularArray));
	gpSpecularArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexSpecular));
}

void BindTransferFunctionRoughness(CTransferFunction& TransferFunctionRoughness)
{
	gTexRoughness.normalized		= true;
	gTexRoughness.filterMode		= hipFilterModeLinear;
	gTexRoughness.addressMode[0]	= hipAddressModeClamp;

	float Roughness[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
		Roughness[i] = TransferFunctionRoughness.F((float)i * INV_TF_NO_SAMPLES).r;
	
	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float>();

	if (gpRoughnessArray == NULL)
		HandleCudaError(hipMallocArray(&gpRoughnessArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpRoughnessArray, 0, 0, Roughness, TF_NO_SAMPLES * sizeof(float),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexRoughness, gpRoughnessArray, ChannelDesc));
}

void UnbindTransferFunctionRoughness(void)
{
	HandleCudaError(hipFreeArray(gpRoughnessArray));
	gpRoughnessArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexRoughness));
}

void BindTransferFunctionEmission(CTransferFunction& TransferFunctionEmission)
{
	gTexEmission.normalized		= true;
	gTexEmission.filterMode		= hipFilterModeLinear;
	gTexEmission.addressMode[0]	= hipAddressModeClamp;

	float4 Emission[TF_NO_SAMPLES];

	for (int i = 0; i < TF_NO_SAMPLES; i++)
	{
		Emission[i].x = TransferFunctionEmission.F((float)i * INV_TF_NO_SAMPLES).r;
		Emission[i].y = TransferFunctionEmission.F((float)i * INV_TF_NO_SAMPLES).g;
		Emission[i].z = TransferFunctionEmission.F((float)i * INV_TF_NO_SAMPLES).b;
	}

	hipChannelFormatDesc ChannelDesc = hipCreateChannelDesc<float4>();
	
	if (gpEmissionArray == NULL)
		HandleCudaError(hipMallocArray(&gpEmissionArray, &ChannelDesc, TF_NO_SAMPLES, 1));

	HandleCudaError(hipMemcpyToArray(gpEmissionArray, 0, 0, Emission, TF_NO_SAMPLES * sizeof(float4),  hipMemcpyHostToDevice));
	HandleCudaError(hipBindTextureToArray(gTexEmission, gpEmissionArray, ChannelDesc));
}

void UnbindTransferFunctionEmission(void)
{
	HandleCudaError(hipFreeArray(gpEmissionArray));
	gpEmissionArray = NULL;
	HandleCudaError(hipUnbindTexture(gTexEmission));
}
#endif

void Vec3ToFloat3(Vec3f* src, float3* dest) {
	dest->x = src->x;
	dest->y = src->y;
	dest->z = src->z;
}
void RGBToFloat3(CColorRgbHdr* src, float3* dest) {
	dest->x = src->r;
	dest->y = src->g;
	dest->z = src->b;
}

void FillCudaCamera(CScene* pScene, CudaCamera& c) {
	Vec3ToFloat3(&pScene->m_Camera.m_From, &c.m_From);
	Vec3ToFloat3(&pScene->m_Camera.m_N, &c.m_N);
	Vec3ToFloat3(&pScene->m_Camera.m_U, &c.m_U);
	Vec3ToFloat3(&pScene->m_Camera.m_V, &c.m_V);
	c.m_ApertureSize = pScene->m_Camera.m_Aperture.m_Size;
	c.m_FocalDistance = pScene->m_Camera.m_Focus.m_FocalDistance;
	c.m_InvScreen[0] = pScene->m_Camera.m_Film.m_InvScreen.x;
	c.m_InvScreen[1] = pScene->m_Camera.m_Film.m_InvScreen.y;
	c.m_Screen[0][0] = pScene->m_Camera.m_Film.m_Screen[0][0];
	c.m_Screen[1][0] = pScene->m_Camera.m_Film.m_Screen[1][0];
	c.m_Screen[0][1] = pScene->m_Camera.m_Film.m_Screen[0][1];
	c.m_Screen[1][1] = pScene->m_Camera.m_Film.m_Screen[1][1];
}

void BindConstants(CScene* pScene, CScene* pDeviceScene)
{
	const float3 AaBbMin = make_float3(pScene->m_BoundingBox.GetMinP().x, pScene->m_BoundingBox.GetMinP().y, pScene->m_BoundingBox.GetMinP().z);
	const float3 AaBbMax = make_float3(pScene->m_BoundingBox.GetMaxP().x, pScene->m_BoundingBox.GetMaxP().y, pScene->m_BoundingBox.GetMaxP().z);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gAaBbMin), &AaBbMin, sizeof(float3)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gAaBbMax), &AaBbMax, sizeof(float3)));

	const float3 InvAaBbMin = make_float3(pScene->m_BoundingBox.GetInvMinP().x, pScene->m_BoundingBox.GetInvMinP().y, pScene->m_BoundingBox.GetInvMinP().z);
	const float3 InvAaBbMax = make_float3(pScene->m_BoundingBox.GetInvMaxP().x, pScene->m_BoundingBox.GetInvMaxP().y, pScene->m_BoundingBox.GetInvMaxP().z);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvAaBbMin), &InvAaBbMin, sizeof(float3)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvAaBbMax), &InvAaBbMax, sizeof(float3)));

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDiffuseColor), pScene->m_DiffuseColor, sizeof(float4)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gSpecularColor), pScene->m_SpecularColor, sizeof(float4)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gEmissiveColor), pScene->m_EmissiveColor, sizeof(float4)));

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gShadingType), &pScene->m_ShadingType, sizeof(int)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGradientFactor), &pScene->m_GradientFactor, sizeof(float)));

	const float IntensityMin		= pScene->m_IntensityRange.GetMin();
	const float IntensityMax		= pScene->m_IntensityRange.GetMax();
	const float IntensityRange		= pScene->m_IntensityRange.GetRange();
	const float IntensityInvRange	= 1.0f / IntensityRange;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gIntensityMin), &IntensityMin, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gIntensityMax), &IntensityMax, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gIntensityRange), &IntensityRange, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gIntensityInvRange), &IntensityInvRange, sizeof(float)));

	const float StepSize		= pScene->m_StepSizeFactor * pScene->m_GradientDelta;
	const float StepSizeShadow	= pScene->m_StepSizeFactorShadow * pScene->m_GradientDelta;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gStepSize), &StepSize, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gStepSizeShadow), &StepSizeShadow, sizeof(float)));

	const float DensityScale = pScene->m_DensityScale;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDensityScale), &DensityScale, sizeof(float)));
	
	const float GradientDelta		= 1.0f * pScene->m_GradientDelta;
	const float InvGradientDelta	= 1.0f / GradientDelta;
	const Vec3f GradientDeltaX(GradientDelta, 0.0f, 0.0f);
	const Vec3f GradientDeltaY(0.0f, GradientDelta, 0.0f);
	const Vec3f GradientDeltaZ(0.0f, 0.0f, GradientDelta);
	
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGradientDelta), &GradientDelta, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvGradientDelta), &InvGradientDelta, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGradientDeltaX), &GradientDeltaX, sizeof(float3)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGradientDeltaY), &GradientDeltaY, sizeof(float3)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGradientDeltaZ), &GradientDeltaZ, sizeof(float3)));
	
	const int FilmWidth		= pScene->m_Camera.m_Film.GetWidth();
	const int Filmheight	= pScene->m_Camera.m_Film.GetHeight();
	const int FilmNoPixels	= pScene->m_Camera.m_Film.m_Resolution.GetNoElements();

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gFilmWidth), &FilmWidth, sizeof(int)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gFilmHeight), &Filmheight, sizeof(int)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gFilmNoPixels), &FilmNoPixels, sizeof(int)));

	const int FilterWidth = 1;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gFilterWidth), &FilterWidth, sizeof(int)));

	const float FilterWeights[10] = { 0.11411459588254977f, 0.08176668094332218f, 0.03008028089187349f, 0.01f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gFilterWeights), FilterWeights, 10 * sizeof(float)));

	const float Gamma		= pScene->m_Camera.m_Film.m_Gamma;
	const float InvGamma	= 1.0f / Gamma;
	const float Exposure	= pScene->m_Camera.m_Film.m_Exposure;
	const float InvExposure	= 1.0f / Exposure;

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gExposure), &Exposure, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvExposure), &InvExposure, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gGamma), &Gamma, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvGamma), &InvGamma, sizeof(float)));

	const float denoiseEnabled = pScene->m_DenoiseParams.m_Enabled ? 1.0f : 0.0f;
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseEnabled), &denoiseEnabled, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseWindowRadius), &pScene->m_DenoiseParams.m_WindowRadius, sizeof(int)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseInvWindowArea), &pScene->m_DenoiseParams.m_InvWindowArea, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseNoise), &pScene->m_DenoiseParams.m_Noise, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseWeightThreshold), &pScene->m_DenoiseParams.m_WeightThreshold, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseLerpThreshold), &pScene->m_DenoiseParams.m_LerpThreshold, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gDenoiseLerpC), &pScene->m_DenoiseParams.m_LerpC, sizeof(float)));

	const float NoIterations	= pScene->GetNoIterations();
	const float InvNoIterations = 1.0f / ((NoIterations > 1.0f) ? NoIterations : 1.0f);

	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gNoIterations), &NoIterations, sizeof(float)));
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvNoIterations), &InvNoIterations, sizeof(float)));

	CudaCamera c;
	FillCudaCamera(pScene, c);
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gCamera), &c, sizeof(CudaCamera)));
	
	CudaLighting cl;
	cl.m_NoLights = pScene->m_Lighting.m_NoLights;
	for (int i = 0; i < cl.m_NoLights; ++i) {
		cl.m_Lights[i].m_Theta = pScene->m_Lighting.m_Lights[i].m_Theta;
		cl.m_Lights[i].m_Phi = pScene->m_Lighting.m_Lights[i].m_Phi;
		cl.m_Lights[i].m_Width = pScene->m_Lighting.m_Lights[i].m_Width;
		cl.m_Lights[i].m_InvWidth = pScene->m_Lighting.m_Lights[i].m_InvWidth;
		cl.m_Lights[i].m_HalfWidth = pScene->m_Lighting.m_Lights[i].m_HalfWidth;
		cl.m_Lights[i].m_InvHalfWidth = pScene->m_Lighting.m_Lights[i].m_InvHalfWidth;
		cl.m_Lights[i].m_Height = pScene->m_Lighting.m_Lights[i].m_Height;
		cl.m_Lights[i].m_InvHeight = pScene->m_Lighting.m_Lights[i].m_InvHeight;
		cl.m_Lights[i].m_HalfHeight = pScene->m_Lighting.m_Lights[i].m_HalfHeight;
		cl.m_Lights[i].m_InvHalfHeight = pScene->m_Lighting.m_Lights[i].m_InvHalfHeight;
		cl.m_Lights[i].m_Distance = pScene->m_Lighting.m_Lights[i].m_Distance;
		cl.m_Lights[i].m_SkyRadius = pScene->m_Lighting.m_Lights[i].m_SkyRadius;
		Vec3ToFloat3(&pScene->m_Lighting.m_Lights[i].m_P, &cl.m_Lights[i].m_P);
		Vec3ToFloat3(&pScene->m_Lighting.m_Lights[i].m_Target, &cl.m_Lights[i].m_Target);
		Vec3ToFloat3(&pScene->m_Lighting.m_Lights[i].m_N, &cl.m_Lights[i].m_N);
		Vec3ToFloat3(&pScene->m_Lighting.m_Lights[i].m_U, &cl.m_Lights[i].m_U);
		Vec3ToFloat3(&pScene->m_Lighting.m_Lights[i].m_V, &cl.m_Lights[i].m_V);
		cl.m_Lights[i].m_Area = pScene->m_Lighting.m_Lights[i].m_Area;
		cl.m_Lights[i].m_AreaPdf = pScene->m_Lighting.m_Lights[i].m_AreaPdf;
		RGBToFloat3(&pScene->m_Lighting.m_Lights[i].m_Color, &cl.m_Lights[i].m_Color);
		RGBToFloat3(&pScene->m_Lighting.m_Lights[i].m_ColorTop, &cl.m_Lights[i].m_ColorTop);
		RGBToFloat3(&pScene->m_Lighting.m_Lights[i].m_ColorMiddle, &cl.m_Lights[i].m_ColorMiddle);
		RGBToFloat3(&pScene->m_Lighting.m_Lights[i].m_ColorBottom, &cl.m_Lights[i].m_ColorBottom);
		cl.m_Lights[i].m_T = pScene->m_Lighting.m_Lights[i].m_T;
	}
	HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gLighting), &c, sizeof(CudaLighting)));

	// copy entire Scene(host mem) up to gpu.
	HandleCudaError(hipMemcpy(pDeviceScene, pScene, sizeof(CScene), hipMemcpyHostToDevice));
}

void Render(const int& Type, CScene& Scene, CScene* pDevScene,
	cudaFB& framebuffers,
	const cudaVolume& volumedata,
	CTiming& RenderImage, CTiming& BlurImage, CTiming& PostProcessImage, CTiming& DenoiseImage)
{
	CScene* pDScene = pDevScene;
//	CScene* pDScene = NULL;
//	HandleCudaError(hipMalloc(&pDScene, sizeof(CScene)));
//	HandleCudaError(hipMemcpy(pDScene, &Scene, sizeof(CScene), hipMemcpyHostToDevice));

	//LOG_DEBUG << "CScene is " << sizeof(CScene) << " bytes";

	// find nearest intersection to set camera focal distance automatically.
	// then re-upload that data.
	if (Scene.m_Camera.m_Focus.m_Type == 0) {
		Scene.m_Camera.m_Focus.m_FocalDistance = NearestIntersection(pDScene, volumedata);
		// send m_FocalDistance back to gpu.
		CudaCamera c;
		FillCudaCamera(&Scene, c);
		HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gCamera), &c, sizeof(CudaCamera)));

		//HandleCudaError(hipMemcpy(pDScene, &Scene, sizeof(CScene), hipMemcpyHostToDevice));
	}

	for (int i = 0; i < Scene.m_Camera.m_Film.m_ExposureIterations; ++i) {
		CCudaTimer TmrRender;

		switch (Type)
		{
		case 0:
		{
			SingleScattering(&Scene, pDScene, volumedata, framebuffers.fb, framebuffers.randomSeeds1, framebuffers.randomSeeds2);
			break;
		}

		case 1:
		{
			//			MultipleScattering(&Scene, pDScene);
			break;
		}
		}
		RenderImage.AddDuration(TmrRender.ElapsedTime());

		// estimate just adds to accumulation buffer.
		CCudaTimer TmrPostProcess;
		Estimate(&Scene, framebuffers.fb, framebuffers.fbaccum);
		PostProcessImage.AddDuration(TmrPostProcess.ElapsedTime());

		Scene.SetNoIterations(Scene.GetNoIterations() + 1);

		const float NoIterations = Scene.GetNoIterations();
		const float InvNoIterations = 1.0f / ((NoIterations > 1.0f) ? NoIterations : 1.0f);
		HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gNoIterations), &NoIterations, sizeof(float)));
		HandleCudaError(hipMemcpyToSymbol(HIP_SYMBOL(gInvNoIterations), &InvNoIterations, sizeof(float)));
	}

//	HandleCudaError(hipFree(pDScene));
//	pDScene = NULL;

}